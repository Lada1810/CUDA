#include "hip/hip_runtime.h"
﻿/***********************LABA3*************************/
//Атор: Русина Лада
//Группа М80-114-М-21
//Дата 11.11.2021
//Вариат 9 
/*-----------------------------------------------------*/


#include "hip/hip_runtime.h"
#include ""
#include "MyHed.h"
#include <thrust/device_vector.h>

#ifndef __HIPCC__ 
#define __HIPCC__
#endif

#include <hip/device_functions.h>


using namespace std;

typedef struct
{
	float val;
	bool indicator = false;
} root;

__global__ void kernel_(root* r, double dx, double start)
{
	__shared__ double temp[3];
	__shared__ double val[3];
	__shared__ bool stop;


	val[threadIdx.x] = (blockIdx.x * dx + threadIdx.x * dx * 0.5) + start;

	if(0.89 * pow(val[threadIdx.x],3) - 2.8 * pow(val[threadIdx.x],2) - 3.7 * val[threadIdx.x] +11.2 == 0)
	{
		r[blockIdx.x].val = 0.;
		r[blockIdx.x].indicator = true;
		stop = true;
	}
	__syncthreads();

	while (!stop)
	{

		if ((log(8 * val[threadIdx.x]) - 9 * val[threadIdx.x] + 3) < 0)
			temp[threadIdx.x] = 0;
		else
			temp[threadIdx.x] = 1;

		__syncthreads();

		if (threadIdx.x == 1)
		{
			if (abs(val[threadIdx.x + 1] - val[threadIdx.x - 1]) < 0.001)
			{
				r[blockIdx.x].val = val[threadIdx.x];
				r[blockIdx.x].indicator = true;

				stop = true;
				break;
			}

			if (temp[threadIdx.x - 1] != temp[threadIdx.x])
			{
				val[threadIdx.x + 1] = val[threadIdx.x];
				val[threadIdx.x] = (val[threadIdx.x - 1] + val[threadIdx.x + 1]) / 2;

			}
			else if (temp[threadIdx.x + 1] != temp[threadIdx.x])
			{
				val[threadIdx.x - 1] = val[threadIdx.x];
				val[threadIdx.x] = (val[threadIdx.x - 1] + val[threadIdx.x + 1]) / 2;
			}
			else
			{
				stop = true;
				break;
			}
		}
	}
}

int main()
{
	double start = 0.0;
	double stop = 10.0;

	int sub_intervals = 100.0;

	double interval_size = stop - start;

	double dx = interval_size / sub_intervals;

	hipEvent_t s;
	hipEvent_t e;

	thrust::device_vector<root> vec(sub_intervals);

	CUDA_CHECK_ERROR(hipEventCreate(&s));
	CUDA_CHECK_ERROR(hipEventCreate(&e));

	CUDA_CHECK_ERROR(hipEventRecord(s, 0));

	hipEvent_t start, stop;
	float gpu_time = 0.0;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);


	kernel_ << <sub_intervals, 3 >> > (thrust::raw_pointer_cast(&vec[0]), dx, start);


	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&gpu_time, start, stop);

	hipEventDestroy(start);
	hipEventDestroy(stop);
	printf("\ntime = %2fmiliseconds;\n", gpu_time);

	printf("Корни уравнения ровняются:\n")
	int roots = 0;
	for (int i = 0; i < vec.size(); i++)
	{
		root r = vec[i];
		if (r.indicator)
		{
			roots++;
			printf("x%d = %f\n", roots, r.val);
		}
	}
	
	CUDA_CHECK_ERROR(hipEventRecord(e, 0));


	CUDA_CHECK_ERROR(hipEventSynchronize(e));

	float time_elapsed = 0;
	CUDA_CHECK_ERROR(hipEventElapsedTime(&time_elapsed, s, e));
	printf("\n");
	printf("done in %f milliseconds\n", time_elapsed);

	return 0;
}