#include "hip/hip_runtime.h"
﻿/***********************LABA3*************************/
//Атор: Русина Лада
//Группа М80-114-М-21
//Дата 11.11.2021
//Вариат 9 
/*-----------------------------------------------------*/



#include <stdio.h>

#include "Header1.h"



/* функция корни которой необходимо найти */
__device__ double CalculateFancshion(double x)
{
	return 0.89 * pow(x,3) - 2.8 * pow(x,2) - 3.7 * x + 11.2;
}



/*функция реализующая метод половинного деления*/
__global__ void kernel_(Calculate_Struct* r_structure, double dx, double start)
{
	__shared__ double temp[3];	__shared__ double val[3];	__shared__ bool stop;	//объявляем 2 промежуточных буфера и флаг конца итераций


	val[threadIdx.x] = (blockIdx.x * dx + threadIdx.x * dx * 0.5) + start;

	if (CalculateFancshion(val[threadIdx.x]) == 0)
	{
		r_structure[blockIdx.x].val = 0.;
		r_structure[blockIdx.x].indicator = true;
		stop = true;
	}
	__syncthreads();//синхронизируем вычисление на нитях

	while (!stop) //запускаем цикл вычисления пока флаг не станет ложным 
	{

		if (CalculateFancshion(val[threadIdx.x]) < 0)
			temp[threadIdx.x] = 0;
		else
			temp[threadIdx.x] = 1;

		__syncthreads();

		if(threadIdx.x == 1)
		{
			if (abs(val[threadIdx.x + 1] - val[threadIdx.x - 1]) < EPSILON)
			{
				r_structure[blockIdx.x].val = val[threadIdx.x];
				r_structure[blockIdx.x].indicator = 1;//true

				stop = 1; //true
				break;
			}

			if (temp[threadIdx.x - 1] != temp[threadIdx.x])
			{
				val[threadIdx.x + 1] = val[threadIdx.x];
				val[threadIdx.x] = (val[threadIdx.x - 1] + val[threadIdx.x + 1]) / 2;

			}
			else if (temp[threadIdx.x + 1] != temp[threadIdx.x])
			{
				val[threadIdx.x - 1] = val[threadIdx.x];
				val[threadIdx.x] = (val[threadIdx.x - 1] + val[threadIdx.x + 1]) / 2;
			}
			else
			{
				stop = false;
				
			}
		}
	}
}


int main()
{
	setlocale(LC_ALL, "ru");
	thrust::device_vector<Calculate_Struct> vec(COUNT_INTERVAl);
	double dx = INTERVAL / COUNT_INTERVAl;

	hipEvent_t start, stop;
	float gpu_time = 0.0;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	kernel_ << <COUNT_INTERVAl, THREADS_PER_BLOCK >> > (thrust::raw_pointer_cast(&vec[0]), dx, START_INTERVAL_RESEARCH);

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&gpu_time, start, stop);

	hipEventDestroy(start);
	hipEventDestroy(stop);
	printf("\ntime = %2fmiliseconds;", gpu_time);


	int index = 0;
	printf("Корни уравнения ровняются:\n")
	for (int i = 0; i < vec.size(); i++)
	{
		Calculate_Struct r = vec[i];
		if (r.indicator)
		{
			index++;
			printf("x %d = %f\n", index, r.val);
		}
	}
	
	


	
	

	return 0;
}