#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <math.h>
#include <locale.h>

#define BLOCK_SIZE 2

__constant__ int Numbers[1024];

void CPU_realisation(int* n)
{
    printf("число %d раскладывается на следующие простые множители\n", *n);
    int a = *n;
    int v = a;
    for (int i = 2; i < a; i++)
    {
        a = v;
        if (a % i == 0)
        {
            //do
            //{
            //	a = a / i;
            printf("i = %d \n ,", i);
            //	if (a / i == 1) { printf("%d",i); }
            //} while (a / i != 1 && a%i == 0);

        }

    }
}
using namespace std;

__global__ void GPU_realization(int* gpu_value)
{
   
    int a = *gpu_value;
    int val = a;
    int i = 2 + threadIdx.x + blockIdx.x * blockDim.x;
    if (i == 2) { printf("the number %d  can be expanded into the following prime factors:\n", *gpu_value); }

    //printf("%d\n", i);
    if (i < a)
    {
        a = val;
        if (a % i == 0)
        {

            do
            {
                a = a / i;
                //printf("%d ,", i);
                Numbers[i-2] = i;
                printf("%d, ", Numbers[i - 2]);
                if (a / i == 1) { Numbers[i] = i; printf("%d ", Numbers[i - 2]); }
            } while (a / i != 1 && a % i == 0);
        }
    }
}

int main()
{
  
   
    
    setlocale(LC_ALL, "ru");
   /* int* value = (int*)malloc(sizeof(int));

    *value = 35;*/
    //CPU_realisation(value);

    int cpu_value = 9;
    int* gpu_value;
    hipMalloc((void**)&gpu_value, sizeof(int));
    hipMemcpy(gpu_value, &cpu_value, sizeof(int), hipMemcpyHostToDevice);

    hipEvent_t start, stop;
    float gpu_time = 0.0;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    GPU_realization << <(cpu_value + 1023) / 1024, 1024 >> > (gpu_value);
    //CPU_realisation(value);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&gpu_time, start, stop);

    hipEventDestroy(start);
    hipEventDestroy(stop);
    printf("\ntime = %2fmiliseconds;", gpu_time);

    

    hipFree(gpu_value);
    return 0;

}