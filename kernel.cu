﻿
#include "hip/hip_runtime.h"


#include <stdio.h>
#include <math.h>
#include <locale.h>

__global__ void expon(double *a, double val)
{
	*a = exp(val);
		
}

void expon_cpu(double *a, double val)
{
	*a = exp(val);
	
}


int main()
{
	setlocale(LC_ALL, "ru");
	printf("В какую степень возвести экспоненту?\n");
	double val;
	scanf("%lf", &val);

	double* a;
	
	double a_cpu;

	hipMalloc((void**)&a, sizeof(double));

	hipEvent_t start, stop;
	float gpu_time = 0.0;
		hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);

	//expon_cpu(&a_cpu, val);												//CPU
	expon <<< 1, 1024 >>> (a, val);											// GPU

	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&gpu_time, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	printf("\nВремя равно: %f", gpu_time);

	
	hipMemcpy(&a_cpu, a, sizeof(double), hipMemcpyDeviceToHost);		//GPU
	printf("\na= %lf", a_cpu);


	hipFree(a);
	
}

